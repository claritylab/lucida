#include "hip/hip_runtime.h"
/* This is the Porter stemming algorithm, coded up as thread-safe ANSI C
  by the author.

  It may be be regarded as cononical, in that it follows the algorithm
  presented in

  Porter, 1980, An algorithm for suffix stripping, Program, Vol. 14,
  no. 3, pp 130-137,

  only differing from it at the points maked --DEPARTURE-- below.

  See also http://www.tartarus.org/~martin/PorterStemmer

  The algorithm as described in the paper could be exactly replicated
  by adjusting the points of DEPARTURE, but this is barely necessary,
  because (a) the points of DEPARTURE are definitely improvements, and
  (b) no encoding of the Porter stemmer I have seen is anything like
  as exact as this version, even with the points of DEPARTURE!

  You can compile it on Unix with 'gcc -O3 -o stem stem.c' after which
  'stem' takes a list of inputs and sends the stemmed equivalent to
  stdout.

  The algorithm as encoded here is particularly fast.

  Release 2 (the more old-fashioned, non-thread-safe version may be
  regarded as release 1.)
*/

#include <stdio.h>
#include <stdlib.h> /* for malloc, free */
#include <ctype.h>  /* for isupper, islower, tolower */
#include <string.h> /* for memcmp, memmove */
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <limits.h>
#include <float.h>
#include <math.h>
#include <sys/time.h>

#include "../../utils/timer.h"

/* You will probably want to move the following declarations to a central
  header file.
*/

struct stemmer;

extern struct stemmer *create_stemmer(void);
extern void free_stemmer(struct stemmer *z);

extern int stem(struct stemmer *z, char *b, int k);

/* The main part of the stemming algorithm starts here.
*/

#define TRUE 1
#define FALSE 0

#define INC 32 /* size units in which s is increased */

/* stemmer is a structure for a few local bits of data,
*/

struct stemmer {
  //  char *b; /* buffer for word to be stemmed */
  char b[INC + 1]; /* buffer for word to be stemmed */
  int k;           /* offset to the end of the string */
  int j;           /* a general offset into the string */
};

/* Member b is a buffer holding a word to be stemmed. The letters are in
  b[0], b[1] ... ending at b[z->k]. Member k is readjusted downwards as
  the stemming progresses. Zero termination is not in fact used in the
  algorithm.

  Note that only lower case sequences are stemmed. Forcing to lower case
  should be done before stem(...) is called.


  Typical usage is:

      struct stemmer * z = create_stemmer();
      char b[] = "pencils";
      int res = stem(z, b, 6);
          /- stem the 7 characters of b[0] to b[6]. The result, res,
             will be 5 (the 's' is removed). -/
      free_stemmer(z);
*/

extern struct stemmer *create_stemmer(void) {
  return (struct stemmer *)malloc(sizeof(struct stemmer));
  /* assume malloc succeeds */
}

extern void free_stemmer(struct stemmer *z) { free(z); }

/* cons(z, i) is TRUE <=> b[i] is a consonant. ('b' means 'z->b', but here
  and below we drop 'z->' in comments.
*/

__host__ __device__ static int cons1(struct stemmer *z, int i) {
  switch (z->b[i]) {
    case 'a':
    case 'e':
    case 'i':
    case 'o':
    case 'u':
      return FALSE;
    default:
      return TRUE;
  }
}

__host__ __device__ static int cons(struct stemmer *z, int i) {
  switch (z->b[i]) {
    case 'a':
    case 'e':
    case 'i':
    case 'o':
    case 'u':
      return FALSE;
    case 'y':
      return (i == 0) ? TRUE : !cons1(z, i - 1);
    default:
      return TRUE;
  }
}

/* m(z) measures the number of consonant sequences between 0 and j. if c is
  a consonant sequence and v a vowel sequence, and <..> indicates arbitrary
  presence,

     <c><v>       gives 0
     <c>vc<v>     gives 1
     <c>vcvc<v>   gives 2
     <c>vcvcvc<v> gives 3
     ....
*/

__host__ __device__ static int m(struct stemmer *z) {
  int n = 0;
  int i = 0;
  int j = z->j;
  while (TRUE) {
    if (i > j) return n;
    if (!cons(z, i)) break;
    i++;
  }
  i++;
  while (TRUE) {
    while (TRUE) {
      if (i > j) return n;
      if (cons(z, i)) break;
      i++;
    }
    i++;
    n++;
    while (TRUE) {
      if (i > j) return n;
      if (!cons(z, i)) break;
      i++;
    }
    i++;
  }
}

/* vowelinstem(z) is TRUE <=> 0,...j contains a vowel */

__host__ __device__ static int vowelinstem(struct stemmer *z) {
  int j = z->j;
  int i;
  for (i = 0; i <= j; i++)
    if (!cons(z, i)) return TRUE;
  return FALSE;
}

/* doublec(z, j) is TRUE <=> j,(j-1) contain a double consonant. */

__host__ __device__ static int doublec(struct stemmer *z, int j) {
  char *b = z->b;
  if (j < 1) return FALSE;
  if (b[j] != b[j - 1]) return FALSE;
  return cons(z, j);
}

/* cvc(z, i) is TRUE <=> i-2,i-1,i has the form consonant - vowel - consonant
  and also if the second c is not w,x or y. this is used when trying to
  restore an e at the end of a short word. e.g.

     cav(e), lov(e), hop(e), crim(e), but
     snow, box, tray.

*/

__host__ __device__ static int cvc(struct stemmer *z, int i) {
  if (i < 2 || !cons(z, i) || cons(z, i - 1) || !cons(z, i - 2)) return FALSE;
  {
    int ch = z->b[i];
    if (ch == 'w' || ch == 'x' || ch == 'y') return FALSE;
  }
  return TRUE;
}

/* ends(z, s) is TRUE <=> 0,...k ends with the string s. */

__host__ __device__ static int memcmp1(const void *buffer1, const void *buffer2,
                                       int count) {
  if (!count) return (0);
  while (--count && *(char *)buffer1 == *(char *)buffer2) {
    buffer1 = (char *)buffer1 + 1;
    buffer2 = (char *)buffer2 + 1;
  }
  return (*((unsigned char *)buffer1) - *((unsigned char *)buffer2));
}

__host__ __device__ static int ends(struct stemmer *z, char *s) {
  int length = s[0];
  char *b = z->b;
  int k = z->k;
  if (s[length] != b[k]) return FALSE; /* tiny speed-up */
  if (length > k + 1) return FALSE;
  if (memcmp1(b + k - length + 1, s + 1, length) != 0) return FALSE;
  z->j = k - length;
  return TRUE;
}

/* setto(z, s) sets (j+1),...k to the characters in the string s, readjusting
  k. */

__host__ __device__ void memmove1(void *dst, const void *src, int count) {
  char *dst_t;
  char *src_t;
  if ((unsigned char *)dst <= (unsigned char *)src ||
      (unsigned char *)dst >= ((unsigned char *)src + count)) {
    dst_t = (char *)dst;
    src_t = (char *)src;
    while (count--) {
      *dst_t++ = *src_t++;
    }
  } else {
    dst_t = (char *)dst + count - 1;
    src_t = (char *)src + count - 1;
    while (count--) {
      *dst_t-- = *src_t--;
    }
  }
}

__host__ __device__ static void setto(struct stemmer *z, char *s) {
  int length = s[0];
  int j = z->j;
  memmove1(z->b + j + 1, s + 1, length);
  z->k = j + length;
}

/* r(z, s) is used further down. */

__host__ __device__ static void r(struct stemmer *z, char *s) {
  if (m(z) > 0) setto(z, s);
}

/* step1ab(z) gets rid of plurals and -ed or -ing. e.g.

      caresses  ->  caress
      ponies    ->  poni
      ties      ->  ti
      caress    ->  caress
      cats      ->  cat

      feed      ->  feed
      agreed    ->  agree
      disabled  ->  disable

      matting   ->  mat
      mating    ->  mate
      meeting   ->  meet
      milling   ->  mill
      messing   ->  mess

      meetings  ->  meet

*/

/* In stem(z, b, k), b is a char pointer, and the string to be stemmed is
  from b[0] to b[k] inclusive.  Possibly b[k+1] == '\0', but it is not
  important. The stemmer adjusts the characters b[0] ... b[k] and returns
  the new end-point of the string, k'. Stemming never increases word
  length, so 0 <= k' <= k.
*/

__host__ __device__ static void step1ab(struct stemmer *z) {
  char *b = z->b;
  if (b[z->k] == 's') {
    if (ends(z,
             "\04"
             "sses"))
      z->k -= 2;
    else if (ends(z,
                  "\03"
                  "ies"))
      setto(z,
            "\01"
            "i");
    else if (b[z->k - 1] != 's')
      z->k--;
  }
  if (ends(z,
           "\03"
           "eed")) {
    if (m(z) > 0) z->k--;
  } else if ((ends(z,
                   "\02"
                   "ed") ||
              ends(z,
                   "\03"
                   "ing")) &&
             vowelinstem(z)) {
    z->k = z->j;
    if (ends(z,
             "\02"
             "at"))
      setto(z,
            "\03"
            "ate");
    else if (ends(z,
                  "\02"
                  "bl"))
      setto(z,
            "\03"
            "ble");
    else if (ends(z,
                  "\02"
                  "iz"))
      setto(z,
            "\03"
            "ize");
    else if (doublec(z, z->k)) {
      z->k--;
      {
        int ch = b[z->k];
        if (ch == 'l' || ch == 's' || ch == 'z') z->k++;
      }
    } else if (m(z) == 1 && cvc(z, z->k))
      setto(z,
            "\01"
            "e");
  }
}

/* step1c(z) turns terminal y to i when there is another vowel in the stem. */

__host__ __device__ static void step1c(struct stemmer *z) {
  if (ends(z,
           "\01"
           "y") &&
      vowelinstem(z))
    z->b[z->k] = 'i';
}

/* step2(z) maps double suffices to single ones. so -ization ( = -ize plus
  -ation) maps to -ize etc. note that the string before the suffix must give
  m(z) > 0. */

__host__ __device__ static void step2(struct stemmer *z) {
  switch (z->b[z->k - 1]) {
    case 'a':
      if (ends(z,
               "\07"
               "ational")) {
        r(z,
          "\03"
          "ate");
        break;
      }
      if (ends(z,
               "\06"
               "tional")) {
        r(z,
          "\04"
          "tion");
        break;
      }
      break;
    case 'c':
      if (ends(z,
               "\04"
               "enci")) {
        r(z,
          "\04"
          "ence");
        break;
      }
      if (ends(z,
               "\04"
               "anci")) {
        r(z,
          "\04"
          "ance");
        break;
      }
      break;
    case 'e':
      if (ends(z,
               "\04"
               "izer")) {
        r(z,
          "\03"
          "ize");
        break;
      }
      break;
    case 'l':
      if (ends(z,
               "\03"
               "bli")) {
        r(z,
          "\03"
          "ble");
        break;
      } /*-DEPARTURE-*/

      /* To match the published algorithm, replace this line with
         case 'l': if (ends(z, "\04" "abli")) { r(z, "\04" "able"); break; } */

      if (ends(z,
               "\04"
               "alli")) {
        r(z,
          "\02"
          "al");
        break;
      }
      if (ends(z,
               "\05"
               "entli")) {
        r(z,
          "\03"
          "ent");
        break;
      }
      if (ends(z,
               "\03"
               "eli")) {
        r(z,
          "\01"
          "e");
        break;
      }
      if (ends(z,
               "\05"
               "ousli")) {
        r(z,
          "\03"
          "ous");
        break;
      }
      break;
    case 'o':
      if (ends(z,
               "\07"
               "ization")) {
        r(z,
          "\03"
          "ize");
        break;
      }
      if (ends(z,
               "\05"
               "ation")) {
        r(z,
          "\03"
          "ate");
        break;
      }
      if (ends(z,
               "\04"
               "ator")) {
        r(z,
          "\03"
          "ate");
        break;
      }
      break;
    case 's':
      if (ends(z,
               "\05"
               "alism")) {
        r(z,
          "\02"
          "al");
        break;
      }
      if (ends(z,
               "\07"
               "iveness")) {
        r(z,
          "\03"
          "ive");
        break;
      }
      if (ends(z,
               "\07"
               "fulness")) {
        r(z,
          "\03"
          "ful");
        break;
      }
      if (ends(z,
               "\07"
               "ousness")) {
        r(z,
          "\03"
          "ous");
        break;
      }
      break;
    case 't':
      if (ends(z,
               "\05"
               "aliti")) {
        r(z,
          "\02"
          "al");
        break;
      }
      if (ends(z,
               "\05"
               "iviti")) {
        r(z,
          "\03"
          "ive");
        break;
      }
      if (ends(z,
               "\06"
               "biliti")) {
        r(z,
          "\03"
          "ble");
        break;
      }
      break;
    case 'g':
      if (ends(z,
               "\04"
               "logi")) {
        r(z,
          "\03"
          "log");
        break;
      } /*-DEPARTURE-*/

      /* To match the published algorithm, delete this line */
  }
}

/* step3(z) deals with -ic-, -full, -ness etc. similar strategy to step2. */

__host__ __device__ static void step3(struct stemmer *z) {
  switch (z->b[z->k]) {
    case 'e':
      if (ends(z,
               "\05"
               "icate")) {
        r(z,
          "\02"
          "ic");
        break;
      }
      if (ends(z,
               "\05"
               "ative")) {
        r(z,
          "\00"
          "");
        break;
      }
      if (ends(z,
               "\05"
               "alize")) {
        r(z,
          "\02"
          "al");
        break;
      }
      break;
    case 'i':
      if (ends(z,
               "\05"
               "iciti")) {
        r(z,
          "\02"
          "ic");
        break;
      }
      break;
    case 'l':
      if (ends(z,
               "\04"
               "ical")) {
        r(z,
          "\02"
          "ic");
        break;
      }
      if (ends(z,
               "\03"
               "ful")) {
        r(z,
          "\00"
          "");
        break;
      }
      break;
    case 's':
      if (ends(z,
               "\04"
               "ness")) {
        r(z,
          "\00"
          "");
        break;
      }
      break;
  }
}

/* step4(z) takes off -ant, -ence etc., in context <c>vcvc<v>. */

__host__ __device__ static void step4(struct stemmer *z) {
  switch (z->b[z->k - 1]) {
    case 'a':
      if (ends(z,
               "\02"
               "al"))
        break;
      return;
    case 'c':
      if (ends(z,
               "\04"
               "ance"))
        break;
      if (ends(z,
               "\04"
               "ence"))
        break;
      return;
    case 'e':
      if (ends(z,
               "\02"
               "er"))
        break;
      return;
    case 'i':
      if (ends(z,
               "\02"
               "ic"))
        break;
      return;
    case 'l':
      if (ends(z,
               "\04"
               "able"))
        break;
      if (ends(z,
               "\04"
               "ible"))
        break;
      return;
    case 'n':
      if (ends(z,
               "\03"
               "ant"))
        break;
      if (ends(z,
               "\05"
               "ement"))
        break;
      if (ends(z,
               "\04"
               "ment"))
        break;
      if (ends(z,
               "\03"
               "ent"))
        break;
      return;
    case 'o':
      if (ends(z,
               "\03"
               "ion") &&
          (z->b[z->j] == 's' || z->b[z->j] == 't'))
        break;
      if (ends(z,
               "\02"
               "ou"))
        break;
      return;
    /* takes care of -ous */
    case 's':
      if (ends(z,
               "\03"
               "ism"))
        break;
      return;
    case 't':
      if (ends(z,
               "\03"
               "ate"))
        break;
      if (ends(z,
               "\03"
               "iti"))
        break;
      return;
    case 'u':
      if (ends(z,
               "\03"
               "ous"))
        break;
      return;
    case 'v':
      if (ends(z,
               "\03"
               "ive"))
        break;
      return;
    case 'z':
      if (ends(z,
               "\03"
               "ize"))
        break;
      return;
    default:
      return;
  }
  if (m(z) > 1) z->k = z->j;
}

/* step5(z) removes a final -e if m(z) > 1, and changes -ll to -l if
  m(z) > 1. */

__host__ __device__ static void step5(struct stemmer *z) {
  char *b = z->b;
  z->j = z->k;
  if (b[z->k] == 'e') {
    int a = m(z);
    if (a > 1 || a == 1 && !cvc(z, z->k - 1)) z->k--;
  }
  if (b[z->k] == 'l' && doublec(z, z->k) && m(z) > 1) z->k--;
}

/* In stem(z, b, k), b is a char pointer, and the string to be stemmed is
  from b[0] to b[k] inclusive.  Possibly b[k+1] == '\0', but it is not
  important. The stemmer adjusts the characters b[0] ... b[k] and returns
  the new end-point of the string, k'. Stemming never increases word
  length, so 0 <= k' <= k.
*/

__global__ void stem_gpu(struct stemmer *stem_list, int words) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < words) {
    if (stem_list[tid].k <= 1) {
      return;
    }

    step1ab(&(stem_list[tid]));
    step1c(&(stem_list[tid]));
    step2(&(stem_list[tid]));
    step3(&(stem_list[tid]));
    step4(&(stem_list[tid]));
    step5(&(stem_list[tid]));
    stem_list[tid].b[stem_list[tid].k + 1] = 0;
  }
}

/*--------------------stemmer definition ends here------------------------*/

#define A_INC 10000

static int i_max = INC; /* maximum offset in s */
struct stemmer *stem_list;
struct stemmer *gpu_stem_list;

#define LETTER(ch) (isupper(ch) || islower(ch))

int load_data(const int WORDS, struct stemmer *stem_list, FILE *f) {
  static int a_max = WORDS;
  int a_size = 0;
  while (TRUE) {
    int ch = getc(f);
    if (ch == EOF) return a_size;
    char *s = (char *)malloc(i_max + 1);
    if (LETTER(ch)) {
      int i = 0;

      while (TRUE) {
        if (i == i_max) {
          i_max += INC;
          s = (char *)realloc(s, i_max + 1);
        }
        ch = tolower(ch); /* forces lower case */

        stem_list[a_size].b[i] = ch;
        s[i] = ch;
        i++;
        ch = getc(f);
        if (!LETTER(ch)) {
          ungetc(ch, f);
          break;
        }
      }

      stem_list[a_size].k = i - 1;
      if (a_size == a_max) {
        a_max += A_INC;
        stem_list = (struct stemmer *)realloc(stem_list,
                                              a_max * sizeof(struct stemmer));
      }
      a_size += 1;
    }
  }
}

int main(int argc, char *argv[]) {
  if (argc < 3) {
    fprintf(stderr, "[ERROR] Invalid arguments provided.\n\n");
    fprintf(stderr, "Usage: %s [WORDS] [INPUT FILE]\n\n", argv[0]);
    exit(0);
  }
  /* Timing */
  STATS_INIT("kernel", "gpu_porter_stemming");
  PRINT_STAT_STRING("abrv", "gpu_stemmer");

  hipEvent_t eStart, eStop;
  float cuda_elapsedTime;

  int WORDS = atoi(argv[1]);
  // allocate data
  FILE *f;
  f = fopen(argv[2], "r");
  if (f == 0) {
    fprintf(stderr, "File %s not found\n", argv[1]);
    exit(1);
  }

  hipHostMalloc((void **)&stem_list, WORDS* sizeof(struct stemmer));

  int words = load_data(WORDS, stem_list, f);
  PRINT_STAT_INT("words", words);

  fclose(f);

  hipEventCreate(&eStart);
  hipEventCreate(&eStop);
  hipMalloc((void **)&gpu_stem_list, words * sizeof(struct stemmer));

  hipEventRecord(eStart, 0);
  hipMemcpy(gpu_stem_list, stem_list, words * sizeof(struct stemmer),
             hipMemcpyHostToDevice);
  hipEventRecord(eStop, 0);
  hipEventSynchronize(eStop);
  hipEventElapsedTime(&cuda_elapsedTime, eStart, eStop);
  PRINT_STAT_DOUBLE("host_to_device", cuda_elapsedTime);

  hipEventRecord(eStart, 0);
  dim3 block(256);
  dim3 grid;
  grid.x = ceil(words * 1.0 / block.x);

  hipEventRecord(eStart, 0);
  stem_gpu << <grid, block>>> (gpu_stem_list, words);
  hipEventRecord(eStop, 0);
  hipEventSynchronize(eStop);
  hipEventElapsedTime(&cuda_elapsedTime, eStart, eStop);
  PRINT_STAT_DOUBLE("gpu_stemmer", cuda_elapsedTime);
  hipEventRecord(eStart, 0);
  hipMemcpy(stem_list, gpu_stem_list, words * sizeof(struct stemmer),
             hipMemcpyDeviceToHost);
  hipEventRecord(eStop, 0);
  hipEventSynchronize(eStop);
  hipEventElapsedTime(&cuda_elapsedTime, eStart, eStop);
  PRINT_STAT_DOUBLE("device_to_host", cuda_elapsedTime);

  hipEventDestroy(eStart);
  hipEventDestroy(eStop);

  STATS_END();
#ifdef TESTING
  f = fopen("../input/stem_porter.gpu", "w");

  for (int i = 0; i < words; ++i) fprintf(f, "%s\n", stem_list[i].b);

  fclose(f);
#endif
  hipHostFree(stem_list);
  hipFree(gpu_stem_list);

  return 0;
}
